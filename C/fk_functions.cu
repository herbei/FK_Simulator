#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>



#define NUM_THREADS 1000
#define NUM_BLOCKS 100


#define HH 1e7

typedef struct {
    int width;
    int height;
    double* elements;
} Matrix;


extern "C" void fkpaths(double *domain, Matrix SITES, Matrix OXY, Matrix UV, double *KXY, Matrix FKSOL);

#define CUDA_CALL(x) do { if ((x) != hipSuccess) { \
printf("Error at %s : %d \n",__FILE__, __LINE__);\
return EXIT_FAILURE;}} while(0)

#define CURAND_CALL(x) do { if ((x) != HIPRAND_STATUS_SUCCESS) { \
printf("Error at %s : %d\n",__FILE__, __LINE__);\
return EXIT_FAILURE;}} while(0)


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}




//###################################################################
__global__ void setup_kernel(hiprandState *state)
{


  int bid=blockIdx.x;
  int tid=threadIdx.x;
  int NPATHS = blockDim.x;


  int thread=bid*NPATHS+tid;

  // each thread gets same seed, different seq number, no offset
  hiprand_init(1234,thread,0,&state[thread]);
}
//##################################################################




__global__ void gpu_fkpaths(hiprandState *state, double *dev_domain, Matrix dev_OXY, Matrix dev_UV, double *dev_KXY, Matrix dev_SITES, Matrix dev_FKSOL){

	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int NPATHS = blockDim.x;


	int thread = bid*NPATHS + tid;

	// copy state to local memory for efficiency;
  	hiprandState localState=state[thread];



  	int M,N;
  	M = dev_OXY.height;
  	N = dev_OXY.width;


	double xc,yc; //x,y current
  	double xn,yn; //x,y new
  	double tau=0.0;

  	double delx;
  	delx=(dev_domain[1]-dev_domain[0])/(double)(N-1);
  
  	double dely;
  	dely=(dev_domain[3]-dev_domain[2])/(double)(M-1);


  	xc = dev_SITES.elements[bid*dev_SITES.width + 0];
  	yc = dev_SITES.elements[bid*dev_SITES.width + 1];

  	int i,j;
  	double uc,vc;

  	tau = 0.0;
  	while(  ((xc - dev_domain[0])*(xc - dev_domain[1])<0) && ((yc - dev_domain[2])*(yc - dev_domain[3])<0)  ){

  		//find uvindx
    	j = ceil( (xc - dev_domain[0])/delx );
    	i = ceil( (yc - dev_domain[2])/dely );
    	if (j<0){
      			j=0;
    		}
    	else{
      			if (j>(N-1)) j=N-1;
    		}
    
    	if (i<0){
      			i=0;
    		}
    	else{
      			if (i>(M-1)) i=M-1;
    		}

	    uc=-dev_UV.elements[i*dev_UV.width+j];
    	vc=-dev_UV.elements[i*dev_UV.width+N+j];


    	xn = xc + HH * uc + sqrt(HH)*sqrt(2*dev_KXY[0])*hiprand_normal(&localState);
    	yn = yc + HH * vc + sqrt(HH)*sqrt(2*dev_KXY[1])*hiprand_normal(&localState);

    	xc=xn;
    	yc=yn;
    	tau = tau + HH;


  	}


  	double lam = 1e-11;
  	int II,JJ;
  	JJ = ceil( (xc - dev_domain[0])/delx );
  	II = ceil( (yc - dev_domain[2])/dely );

  	if (JJ<0){
    		JJ=0;
  		}
  	else{
    		if (JJ>(N-1)) JJ=N-1;
  		}

  	if (II<0){
    		II=0;
  		}
  	else{
    		if (II>(M-1)) II=M-1;
  		}



  	dev_FKSOL.elements[bid * dev_FKSOL.width + tid] = dev_OXY.elements[II * dev_OXY.width + JJ] * exp(-lam*tau);
	//dev_FKSOL.elements[bid * dev_FKSOL.width + tid] = (double)bid;


  	// copy state back to global memory
  	state[thread]=localState;

}




void fkpaths(double *domain, Matrix SITES, Matrix OXY, Matrix UV, double *KXY, Matrix FKSOL){


	int NOBS;
	int NPATHS;

	NOBS = FKSOL.height;
	NPATHS = FKSOL.width;


 	double *dev_domain;
  	gpuErrchk( hipMalloc( (void **)&dev_domain, 4*sizeof(double)) );
  	gpuErrchk( hipMemcpy(dev_domain, domain, 4*sizeof(double), hipMemcpyHostToDevice) );
  	
	Matrix dev_OXY;
  	dev_OXY.height=OXY.height;
  	dev_OXY.width=OXY.width;
  	gpuErrchk( hipMalloc( (void **)&dev_OXY.elements, dev_OXY.height*dev_OXY.width*sizeof(double)) );
  	gpuErrchk( hipMemcpy(dev_OXY.elements, OXY.elements, dev_OXY.height*dev_OXY.width*sizeof(double), hipMemcpyHostToDevice) );

  	Matrix dev_UV;
	dev_UV.height=UV.height;
  	dev_UV.width=UV.width;
  	gpuErrchk( hipMalloc( (void **)&dev_UV.elements, dev_UV.height*dev_UV.width*sizeof(double)) );
  	gpuErrchk( hipMemcpy(dev_UV.elements, UV.elements, dev_UV.height*dev_UV.width*sizeof(double), hipMemcpyHostToDevice) );

 	double *dev_KXY;
  	gpuErrchk( hipMalloc( (void **)&dev_KXY, 2*sizeof(double)) );
  	gpuErrchk( hipMemcpy(dev_KXY, KXY, 2*sizeof(double), hipMemcpyHostToDevice) );


  	Matrix dev_SITES;
  	dev_SITES.height = SITES.height;
  	dev_SITES.width = SITES.width;
  	gpuErrchk( hipMalloc( (void **)&dev_SITES.elements, dev_SITES.height*dev_SITES.width*sizeof(double) ) );
  	gpuErrchk( hipMemcpy(dev_SITES.elements, SITES.elements, dev_SITES.height * dev_SITES.width * sizeof(double), hipMemcpyHostToDevice) );

	Matrix dev_FKSOL;
  	dev_FKSOL.height=NOBS; 
  	dev_FKSOL.width=NPATHS;
  	gpuErrchk( hipMalloc( (void **)&dev_FKSOL.elements, dev_FKSOL.height*dev_FKSOL.width*sizeof(double)) );
	gpuErrchk( hipPeekAtLastError() );
    
  	



  	hiprandState *devStates;
  	gpuErrchk ( hipMalloc( (void **)&devStates, NPATHS*sizeof(hiprandState)) );
  	setup_kernel<<<NOBS, NPATHS>>>(devStates);
	gpuErrchk( hipPeekAtLastError() );
	

  	gpu_fkpaths<<<NOBS,NPATHS>>>(devStates, dev_domain, dev_OXY, dev_UV, dev_KXY, dev_SITES, dev_FKSOL);
	gpuErrchk( hipPeekAtLastError() );

	gpuErrchk( hipMemcpy(FKSOL.elements, dev_FKSOL.elements, FKSOL.height * FKSOL.width * sizeof(double), hipMemcpyDeviceToHost) );
	gpuErrchk( hipPeekAtLastError() );
	//gpuErrchk( cudaDeviceSynchronize() );



	//printf("test = %f\n",FKSOL.elements[3]);

  	//free
  	hipFree(dev_domain);
	hipFree(dev_OXY.elements);
	hipFree(dev_UV.elements);
	hipFree(dev_KXY);
	hipFree(dev_SITES.elements);
	hipFree(dev_FKSOL.elements);



	//printf("%s\n", "done.");
}





